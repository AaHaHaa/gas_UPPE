
#include <hip/hip_runtime.h>
__global__ void get_GNLSE_nonlinearity_air(double2* PK, double2* P_p12,
                                            const double2* A_t,
                                            const double permittivity0,
                                            const double Ng_N2,
                                            const double B_N2, const double2* p12_R_N2,
                                            const double a_N2, const double2* p12_V_N2,
                                            const double Ng_O2,
                                            const double B_O2, const double2* p12_R_O2,
                                            const double a_O2, const double2* p12_V_O2,
                                            const unsigned int N,
                                            const double* DW) {
    const unsigned int thread_idx = threadIdx.x + blockIdx.x*blockDim.x;
    const unsigned int parallel_num = 10; // the number of parallelization
    const unsigned int Nf_in_block = 102;// floor(MaxThreadsPerBlock / parallel_num)
    const unsigned int Ni_in_block = threadIdx.x / parallel_num;
    const unsigned int parallel_idx = thread_idx % parallel_num;
    const unsigned int Ni = thread_idx / parallel_num;

    if (thread_idx >= parallel_num*N) return;

    __shared__ double I[Nf_in_block];
    __shared__ double2 A2[Nf_in_block];

    const double2 A = A_t[Ni];
    /* 
    *  I = abs(A)^2 + abs(A_p)^2
    *  A2 = A^2 + A_p^2
    */
    if (parallel_idx == 0) {
        I[Ni_in_block] = pow(A.x,2) + pow(A.y,2);
            A2[Ni_in_block].x = pow(A.x,2) - pow(A.y,2);
            A2[Ni_in_block].y = 2*A.x*A.y;
    }
    __syncthreads();

    // Kerr
    double2 conjA; conjA.x = A.x; conjA.y = -A.y;
    double2 K1;
    // Raman
    __shared__ double2 this_p12_R_N2[Nf_in_block];
    __shared__ double2 this_p12_V_N2[Nf_in_block];
    __shared__ double2 this_p12_R_O2[Nf_in_block];
    __shared__ double2 this_p12_V_O2[Nf_in_block];
    switch (parallel_idx) {
        // N2's R
        case 0: // p12_R_N2
            this_p12_R_N2[Ni_in_block] = p12_R_N2[Ni];
            break;

        // N2's V
        case 1: // p12_V_N2
            this_p12_V_N2[Ni_in_block] = p12_V_N2[Ni];
            break;

        // O2's R
        case 2: // p12_R_O2
            this_p12_R_O2[Ni_in_block] = p12_R_O2[Ni];
            break;

        // O2's V
        case 3: // p12_V_O2
            this_p12_V_O2[Ni_in_block] = p12_V_O2[Ni];
            break;
    }
    __syncthreads();

    __shared__ double2 PR_p12_N2[Nf_in_block];
    __shared__ double2 PV_p12_N2[Nf_in_block];
    __shared__ double2 PR_p12_O2[Nf_in_block];
    __shared__ double2 PV_p12_O2[Nf_in_block];

    switch (parallel_idx) {
        /* Kerr term */
        // X3 is taken out because it can be the function of frequency, so it's considered later.
        case 0: // x
            K1.x = conjA.x*A2[Ni_in_block].x - conjA.y*A2[Ni_in_block].y;
            PK[Ni].x = permittivity0/4*(K1.x + 2*A.x*I[Ni_in_block]);
            break;
        case 1: // y
            K1.y = conjA.x*A2[Ni_in_block].y + conjA.y*A2[Ni_in_block].x;
            PK[Ni].y = permittivity0/4*(K1.y + 2*A.y*I[Ni_in_block]);
            break;

        /* stimulated Raman term: N2's R */
        case 2:
            PR_p12_N2[Ni_in_block].x = Ng_N2*B_N2*(2*this_p12_R_N2[Ni_in_block].x)*A.x;
            break;
        case 3:
            PR_p12_N2[Ni_in_block].y = Ng_N2*B_N2*(2*this_p12_R_N2[Ni_in_block].x)*A.y;
            break;

        /* stimulated Raman term: N2's V */
        case 4:
            PV_p12_N2[Ni_in_block].x = Ng_N2*a_N2*(2*this_p12_V_N2[Ni_in_block].x)*A.x;
            break;
        case 5:
            PV_p12_N2[Ni_in_block].y = Ng_N2*a_N2*(2*this_p12_V_N2[Ni_in_block].x)*A.y;
            break;

        /* stimulated Raman term: O2's R */
        case 6:
            PR_p12_O2[Ni_in_block].x = Ng_O2*B_O2*(2*this_p12_R_O2[Ni_in_block].x)*A.x;
            break;
        case 7:
            PR_p12_O2[Ni_in_block].y = Ng_O2*B_O2*(2*this_p12_R_O2[Ni_in_block].x)*A.y;
            break;

        /* stimulated Raman term: O2's V */
        case 8:
            PV_p12_O2[Ni_in_block].x = Ng_O2*a_O2*(2*this_p12_V_O2[Ni_in_block].x)*A.x;
            break;
        case 9:
            PV_p12_O2[Ni_in_block].y = Ng_O2*a_O2*(2*this_p12_V_O2[Ni_in_block].x)*A.y;
            break;
    }
    __syncthreads();

    // Sum Raman terms up
    switch (parallel_idx) {
        /* p12 */
        case 0:
            P_p12[Ni].x = (PR_p12_N2[Ni_in_block].x + PV_p12_N2[Ni_in_block].x + \
                           PR_p12_O2[Ni_in_block].x + PV_p12_O2[Ni_in_block].x   )*DW[Ni];
            break;
        case 1:
            P_p12[Ni].y = (PR_p12_N2[Ni_in_block].y + PV_p12_N2[Ni_in_block].y + \
                           PR_p12_O2[Ni_in_block].y + PV_p12_O2[Ni_in_block].y   )*DW[Ni];
            break;
    }
}